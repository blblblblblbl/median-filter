#include "hip/hip_runtime.h"
#include "stuff.h"

#define TILE_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

//Texture memory utilization
texture<float, 2, hipReadModeElementType> imageTextureRef;

__global__ void medianFilterTrueGPU(float *res, int height, int width) {
    float filter[offset_];
    pairs offset_range[offset_];

    for (int i = -offset; i <= offset; i++) {
        for (int j = -offset; j <= offset; j++) {
            offset_range[(i + offset) * (2 * offset + 1) + j + offset] = {i, j};
        }
    }

    //Parallelized processing for each pixel
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    float u = 0, v= 0, pixel;
    int count = 0;

    //Choose whether if on the edge of the image
    for (int k = 0; k < offset_; k++) {
        pairs p = offset_range[k];
        if (row + p.fi < height && row + p.fi >= 0&&
            col + p.se >= 0 && col + p.se < width) {
            u = (row + p.fi);
            v = (col + p.se);
        } else {
            u = row;
            v = col;
        }
        pixel = tex2D(imageTextureRef, v / (float) width, u / (float) height) ;
        //    printf("%i\n", pixel);
        filter[count++] = pixel;
    }

    //Choose median() for 3x3 matrix around each pixel
    for (int k = 0; k < offset_; k++) {
        for (int k2 = k + 1; k2 < offset_; k2++) {
            if (filter[k] > filter[k2]) {
                auto tmp = filter[k];
                filter[k] = filter[k2];
                filter[k2] = tmp;
            }
        }
    }

    res[row * width + col] = filter[offset_ / 2];

    __syncthreads();
}

static int iterCount = 1;

void MedianFilterGPU(float *res, float *pixel_colors, int height, int width) {
    printf("Iteration :: %i\n", iterCount++);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
            hipCreateChannelDesc(32, 0, 0, 0,
                                  hipChannelFormatKindFloat);
    hipArray *cuArray;
    gpuErrchk(hipMallocArray(&cuArray, &channelDesc, width, height));

    // Copy to device memory some data located at address h_data
    // in host memory
    int sizeBMP = width * height * sizeof(float);
    gpuErrchk(hipMemcpy2DToArray(cuArray, 0, 0, pixel_colors, width * sizeof(float),
                                  width * sizeof(float), height, hipMemcpyHostToDevice));

    // Set texture parameters
//    imageTextureRef.normalized = 0;
    imageTextureRef.addressMode[0] = hipAddressModeWrap;
    imageTextureRef.addressMode[1] = hipAddressModeWrap;
    imageTextureRef.filterMode = hipFilterModeLinear;
    imageTextureRef.normalized = true;

    // Bind the array to the texture reference
    gpuErrchk(hipBindTextureToArray(imageTextureRef, cuArray, channelDesc));

    float *result_device;
    gpuErrchk(hipMalloc(&result_device, sizeBMP));

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((int) ceil(width / (float) TILE_SIZE),
                 (int) ceil((height / (float) TILE_SIZE)));

    gpuErrchk(hipEventRecord(start, 0));
    medianFilterTrueGPU<<<dimGrid, dimBlock>>>(result_device, height, width);

    gpuErrchk(hipMemcpy(res, result_device, sizeBMP, hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time consumed for current kernel call :: %3.1f ms \n", milliseconds);

    // Free device memory
    hipFreeArray(cuArray);
    hipFree(result_device);
    hipDeviceSynchronize();
}